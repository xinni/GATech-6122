
#include <hip/hip_runtime.h>
#include <stdio.h>
                                                                                                                                         
#define N 1029 
#define T_P_B 512

__global__ void double_it(int *a, int *b, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n) {
        b[idx] = 2 * a[idx];
    }
}

int main() {
    int *d_a, *d_b;

    hipMalloc((void**)&d_a, N*sizeof(N));
    hipMalloc((void**)&d_b, N*sizeof(N));
 
    int a[N], b[N];

    for(int i = 0; i < N; ++i) {
        a[i] = i;
    }
 
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
 
    double_it<<<(N + T_P_B-1) / T_P_B, T_P_B>>>(d_a, d_b, N);
 
    hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < N; ++i) {
        printf("%d x2 = %d\n", a[i], b[i]);
    }
 
    hipFree(d_a);
    hipFree(d_b);
 }