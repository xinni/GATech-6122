#include <stdio.h>
#include <iostream>
#include <string>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>
#include <sstream>

#define DIM 32

const float PI = 3.14159265358979f;

using namespace std;

/*************************************************/
class Complex {
public:
    __host__ __device__ Complex() : real(0.0f), imag(0.0f) {}
    
    __host__ __device__ Complex(float r) : real(r), imag(0.0f) {}
    
    __host__ __device__ Complex(float r, float i) : real(r), imag(i) {}
    
    __host__ __device__ Complex operator+(const Complex &b) const {
        float newReal = real + b.real;
        float newImag = imag + b.imag;
        Complex newComplex(newReal, newImag);
        return newComplex;
    }
    
    __host__ __device__ Complex operator-(const Complex &b) const {
        float newReal = real - b.real;
        float newImag = imag - b.imag;
        Complex newComplex(newReal, newImag);
        return newComplex;
    }
    
    __host__ __device__ Complex operator*(const Complex &b) const {
        float newReal = real * b.real - imag * b.imag;
        float newImag = real * b.imag + imag * b.real;
        Complex newComplex(newReal, newImag);
        return newComplex;
    }
    
    __host__ __device__ Complex mag() const {
        float magNum = sqrt(real * real + imag * imag);
        Complex magComplex(magNum);
        return magComplex;
    }
    
    __host__ __device__ Complex angle() const {
        float angle = atan(1.0 * imag / real)*180/PI;
        Complex angleComplex(angle);
        return angleComplex;
    }
    
    __host__ __device__ Complex conj() const {
        Complex newComplex(real, -1.0 * imag);
        return newComplex;
    }

    float real;
    float imag;
};

std::ostream& operator<< (std::ostream& os, const Complex& rhs) {
    Complex c(rhs);
    if(fabsf(rhs.imag) < 1e-10) c.imag = 0.0f;
    if(fabsf(rhs.real) < 1e-10) c.real = 0.0f;

    if(c.imag == 0) {
        os << c.real;
    }
    else {
        os << "(" << c.real << "," << c.imag << ")";
    }
    return os;
}

class InputImage {
public:

    InputImage(const char* filename){
        std::ifstream ifs(filename);
        if(!ifs) {
            std::cout << "Can't open image file " << filename << std::endl;
            exit(1);
        }
    
        ifs >> w >> h;
        data = new Complex[w * h];
        for(int r = 0; r < h; ++r) {
            for(int c = 0; c < w; ++c) {
                // float real;
                // ifs >> real;
                // data[r * w + c] = Complex(real);
                string word;
                ifs >> word;
                int found = word.find_first_not_of(" \t");
                if (word[found] == '(') {
                    istringstream iss(word);
                    char temp;
                    float real, imag;
                    iss >> temp >> real >> temp >> imag >> temp;
                    data[r * w + c] = Complex(real, imag);
                } else {
                    istringstream iss(word);
                    float real;
                    iss >> real;
                    data[r * w + c] = Complex(real);
                }
            }
        }
    }
    int get_width() const{
        return w;
    }
    int get_height() const{
        return h;
    }

    //returns a pointer to the image data.  Note the return is a 1D
    //array which represents a 2D image.  The data for row 1 is
    //immediately following the data for row 0 in the 1D array
    Complex* get_image_data() const{
        return data;
    }
    //use this to save output from forward DFT
    void save_image_data(const char* filename, Complex* d, int w, int h){
        std::ofstream ofs(filename);
        if(!ofs) {
            std::cout << "Can't create output image " << filename << std::endl;
            return;
        }
    
        ofs << w << " " << h << std::endl;
    
        for(int r = 0; r < h; ++r) {
            for(int c = 0; c < w; ++c) {
                ofs << d[r * w + c] << " ";
            }
            ofs << std::endl;
        }
    }
    //use this to save output from reverse DFT
    void save_image_data_real(const char* filename, Complex* d, int w, int h){
        std::ofstream ofs(filename);
        if(!ofs) {
            std::cout << "Can't create output image " << filename << std::endl;
            return;
        }
    
        ofs << w << " " << h << std::endl;
    
        for (int r = 0; r < h; ++r) {
            for (int c = 0; c < w; ++c) {
                ofs << d[r * w + c].real << " ";
            }
            ofs << std::endl;
        }
    }

private:
    int w;
    int h;
    Complex* data;
};
/*************************************************/

struct DeviceData {
    Complex *d_data;
    Complex *d_temp;
    Complex *d_res;
};

void cleanup(DeviceData *d) {
    hipFree(d->d_data);
    hipFree(d->d_temp);
    hipFree(d->d_res);
}

/*************** forward transform by row **********************/
__global__ void transByRow (Complex* dst, Complex* src, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = x + y * width;

    if (x < width && y < height) {
        for (int i = 0; i < width; i++) {
            float re = (src + y*width + i)->real;
            float im = (src + y*width + i)->imag;
            Complex w = Complex(cos(2*PI*i*x/width), -sin(2*PI*i*x/width));
            (dst + index)->real += re * w.real - im*w.imag;
            (dst + index)->imag += re * w.imag + im*w.real;
        }
    }
}

/*************** forward transform by column **********************/
__global__ void transByCol (Complex* dst, Complex* src, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = x + y * width;

    if (x < width && y < height) {
        for (int i = 0; i < height; i++) {
            float re = (src + x + i*width)->real;
            float im = (src + x + i*width)->imag;
            Complex w = Complex(cos(2*PI*i*y/height), -sin(2*PI*i*y/height));
            (dst + index)->real += re * w.real - im*w.imag;
            (dst + index)->imag += re * w.imag + im*w.real;
        }
    }
}

/*************** reverse transform by row **********************/
__global__ void revByRow (Complex* dst, Complex* src, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = x + y * width;

    if (x < width && y < height) {
        for (int i = 0; i < width; i++) {
            float re = (src + y*width + i)->real;
            float im = (src + y*width + i)->imag;
            Complex w = Complex(cos(2*PI*i*x/width), sin(2*PI*i*x/width));
            (dst + index)->real += (re * w.real - im*w.imag)/width;
            (dst + index)->imag += (re * w.imag + im*w.real)/width;
        }
    }
}

/*************** reverse transform by column **********************/
__global__ void revByCol (Complex* dst, Complex* src, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = x + y * width;

    if (x < width && y < height) {
        for (int i = 0; i < height; i++) {
            float re = (src + x + i*width)->real;
            float im = (src + x + i*width)->imag;
            Complex w = Complex(cos(2*PI*i*y/height), sin(2*PI*i*y/height));
            (dst + index)->real += (re * w.real - im*w.imag)/height;
            (dst + index)->imag += (re * w.imag + im*w.real)/height;
        }
    }
}


int main (int argc, char* argv[]) {
    DeviceData devs;

    string str = "forward";
    bool forward = (strcmp(argv[1], str.c_str()) == 0 );
    char* inFile = argv[2];
    char* outFile = argv[3];

    InputImage image(inFile);
    int height = image.get_height();
    int width = image.get_width();
    int N = height * width;

    Complex res[N];
    fill_n(res, N, 1);

    Complex* data = image.get_image_data();

    hipMalloc((void**)&devs.d_data, N * sizeof(Complex));
    hipMalloc((void**)&devs.d_res, N * sizeof(Complex));
    hipMalloc((void**)&devs.d_temp, N * sizeof(Complex));
    hipMemcpy(devs.d_data, data, N * sizeof(Complex), hipMemcpyHostToDevice);

    dim3 blocks((width + DIM - 1) / DIM, (height + DIM - 1) / DIM);
    dim3 threads(DIM, DIM);
    cout << width << ", " << height << forward << endl;

    if (forward) {
        transByRow<<<blocks, threads>>>(devs.d_temp, devs.d_data, width, height);
        transByCol<<<blocks, threads>>>(devs.d_res, devs.d_temp, width, height);

        hipMemcpy(res, devs.d_res, N*sizeof(Complex), hipMemcpyDeviceToHost);
        image.save_image_data(outFile, res, width, height);

    } else {
        revByRow<<<blocks, threads>>>(devs.d_temp, devs.d_data, width, height);
        revByCol<<<blocks, threads>>>(devs.d_res, devs.d_temp, width, height);

        hipMemcpy(res, devs.d_res, N*sizeof(Complex), hipMemcpyDeviceToHost);
        image.save_image_data_real(outFile, res, width, height);
    }

    cleanup(&devs);

    return 0;
}
