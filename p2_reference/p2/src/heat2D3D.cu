#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <stdlib.h>
#include <sstream>
#include <fstream>

#define DIM 8

using namespace std;

struct DeviceData {
    float *d_fix;
    float *d_cur;
    float *d_pre;
    float *in;
    float *out;
};

void cleanup(DeviceData *d ) {
    hipFree(d->d_fix);
    hipFree(d->d_cur);
    hipFree(d->d_pre);
    hipFree(d->in);
    hipFree(d->out);
}


void setInitHeatMap(float *dst, int width, int height, int location_x, int location_y, int widthFix, int heightFix, float fixedTemp) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            if ((i >= location_y) && (i < location_y + heightFix)) {
                if ((j >= location_x) && (j < location_x + widthFix)){
                    dst[i * width + j] = fixedTemp;
                }
            }
        }
    }
}


void setInitHeatMap(float *dst, int width, int height, int depth, int location_x, int location_y, int location_z, int widthFix, int heightFix, int depthFix, float fixedTemp) {
    for (int k = 0; k < depth; k++) {
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                if ((k >= location_z) && (k < location_z + depthFix)) {
                    if ((i >= location_y) && (i < location_y + heightFix)) {
                        if ((j >= location_x) && (j < location_x + widthFix)){
                            dst[k * width * height + i * width + j] = fixedTemp;
                        }
                    }
                }
            }
        }
    }
}


__global__ void copy_const_kernel (float *dst, const float *src, int width, int height, int depth) {
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;

    int index = x + y * width + z * width * height;

    if ((x < width && y < height && z < depth) && (src[index] != 0)) dst[index] = src[index];
}


__global__ void update_kernel (float *dst, float *src, int width, int height, int depth, float k) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    
    int index = x + y * width + z * width * height;

    int left = index - 1;  
    int right = index + 1;  
    if (x == 0) left++;  
    if (x == width - 1) right--;
  
    int top = index - width;
    int bottom = index + width;
    if (y == 0) top += width;
    if (y == height - 1) bottom -= width;

    int front = index - width * height;
    int back = index + width * height;
    if (z == 0) front = front + width * height;
    if (z == depth - 1) back = back - width * height;
  
    if (x < width && y < height && z < depth) {
        dst[index] = src[index] + k * 
            (src[top] + src[bottom] + src[left] + src[right] + src[front] + src[back] - src[index] * 6);
    }
}

int main (int argc, char *argv[]) {
    DeviceData data;

    
    bool dim2D = true; 
    float k, startTemp, *fix;
    int timeSteps, N, width, height, depth = 1;
    
    
    ifstream infile(argv[1]);
    string line;
    int index = 0;
    while (getline(infile, line)) {

        
        int found= line.find_first_not_of(" \t");
        if(found != string::npos) {
            if(line[found] == '#') continue;
        } else {
            continue;
        }
        istringstream iss(line);
        char comma;
        
        
        if (index == 0) {
            
            if (line == "3D") dim2D = false; 

        } else if (index == 1) {
            
            if (!(iss >> k)) break; 

        } else if (index == 2) {
            
            if (!(iss >> timeSteps)) break;

        } else if (index == 3) {
            
            if (dim2D) {
                if (!(iss >> width >> comma >> height) || (comma != ',')) break;
                
            } else {
                if (!(iss >> width >> comma >> height >> comma >> depth) || (comma != ',')) break;
            }
            N = width * height * depth;
            fix = new float[N];
            fill_n(fix, N, 0);

        } else if (index == 4) {
            
            if (!(iss >> startTemp)) break;

        } else {
            
            if (dim2D) {
                int _x, _y, wf, hf;
                float tf;
                if (!(iss >> _x >> comma >> _y >> comma >> wf >> comma >> hf >> 
                    comma >> tf) || (comma != ',')) break;
                setInitHeatMap(fix, width, height, _x, _y, wf, hf, tf);
            } else {
                int _x, _y, _z, wf, hf, df;
                float tf;
                if (!(iss >> _x >> comma >> _y >> comma >> _z >> comma >> wf >> comma >> hf >> 
                    comma >> df >> comma >> tf) || (comma != ',')) break;
                setInitHeatMap(fix, width, height, depth, _x, _y, _z, wf, hf, df, tf);
            }
        }
        index++;
    }
    

    float previous[N];
    float current[N] = {0};
    fill_n(previous, N, startTemp);

    hipMalloc((void**)&data.d_cur, N * sizeof(float));
    hipMalloc((void**)&data.d_pre, N * sizeof(float));
    hipMalloc((void**)&data.d_fix, N * sizeof(float));

    dim3 blocks((width + DIM - 1) / DIM, (height + DIM - 1) / DIM, (depth + DIM - 1) / DIM);
    dim3 threads(DIM, DIM, DIM);

    hipMemcpy(data.d_fix, fix, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data.d_pre, previous, N*sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i <= timeSteps; i++) {
        if (i % 2) {
            data.in = data.d_cur;
            data.out = data.d_pre;
        } else {
            data.in = data.d_pre;
            data.out = data.d_cur;
        }
        
        update_kernel<<<blocks, threads>>>(data.out, data.in, width, height, depth, k);
        copy_const_kernel<<<blocks, threads>>>(data.out, data.d_fix, width, height, depth);
    }

    hipMemcpy(current, data.out, N*sizeof(int), hipMemcpyDeviceToHost);

    
	ofstream outFile;
	outFile.open("heatOutput.csv", ios::out);
    for (int i = 0; i < N; i++) {
        if (i % (width * height) != width * height - 1) {
            if (i % width != width - 1) {
                outFile << current[i] << ", ";
            } else {
                outFile << current[i] << endl;
            }
        } else {
            if (i == N - 1) {
                outFile << current[i] << endl;
            } else {
                outFile << current[i] << endl << endl;
            }
        }
    }
    
    cleanup(&data);

    return 0;
}